#include "hip/hip_runtime.h"
%%cu




#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024
#define N 500000 //50000

using namespace std;


__global__ void sum( int *A,  int stride, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
  
   if(i < stride && i + stride < n)
   A[i] = A[i]+ A[i+stride];
}

int main(void)
{
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;
    int threadsPerBlock;
    int blocksPerGrid; 
    int n;
   int stride;

    threadsPerBlock = BLOCK_SIZE;
    n = N;
 
    blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    
    hipError_t err = hipSuccess;


    int *A = (int *)malloc(n * sizeof(int));
    int result=0;

    for (int i = 0; i < n; ++i)
    {
        A[i] = i+1;
        result = result + A[i];
    }
    
    int *dA = NULL;
    err = hipMalloc((void **)&dA, n * sizeof(int));
   
    err = hipMemcpy(dA, A, n * sizeof(int), hipMemcpyHostToDevice);



    
    hipEventRecord(start,0);
 for(stride=ceil(float(n)/2); stride>=1; stride = ceil(float(stride)/2))
 {
     
      blocksPerGrid = (ceil(float(n)/2) + threadsPerBlock - 1) / threadsPerBlock;
      sum<<<blocksPerGrid, threadsPerBlock>>>(dA, stride, n);
      n = ceil(float(n)/2);
      if(stride==1) break;
 }
    hipEventRecord(stop,0);
    hipEventSynchronize (stop); 
    hipEventElapsedTime (&time, start, stop); 
    
    hipEventDestroy (start);
    hipEventDestroy (stop);
 
     cout<<"Time taken: "<<time<<" milli seconds"<<endl;
    
    err = hipMemcpy(A, dA, sizeof(int), hipMemcpyDeviceToHost);   
    hipDeviceSynchronize();
    cout<<A[0]<<endl;
    cout<<result<<endl;
    
    err = hipFree(dA);    

    free(A);
    

    return 0;
}