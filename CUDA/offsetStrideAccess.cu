#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#define N 1024*1024 
#define BS 1024;
//1M elements 

__global__ void offsetAccess(int *A, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  A[i] = A[i] + 1;
}

__global__ void strideAccess(int *A, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  A[i] = A[i] + 1;
}

void offsetExperiment()
{
  int blockSize,  *dA, n;
  float time;;
  hipEvent_t startEvent, stopEvent;
    
  n = N;  blockSize= BS;


  hipMalloc(&dA, n * 32 * sizeof(int));
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  printf("#Bytes Requested = %d\n", n *4);
  printf("Offset -  Running time (ms)\n");
  
  for (int i = 1; i <= 32; i++) 
  {

    hipEventRecord(startEvent,0);
    offsetAccess<<<n/blockSize, blockSize>>>(dA, i);
    hipEventRecord(stopEvent,0);
    hipEventSynchronize(stopEvent);

    hipEventElapsedTime(&time, startEvent, stopEvent);
  
    printf(" %d %f  \n", i, time);

  }
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
  hipFree(dA);
}

void strideExperiment()
{
  int blockSize,  *dA, n;
  float time;;
  hipEvent_t startEvent, stopEvent;
    
  n = N;  blockSize= BS;


  hipMalloc(&dA, n * 32 * sizeof(int));
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);

  printf("#Bytes Requested = %d\n", n *4);

  printf("Stride - Running time (ms)\n");

  
  for (int i = 1; i <= 32; i++) 
  {
    hipEventRecord(startEvent,0);
   
    strideAccess<<<n/blockSize, blockSize>>>(dA, i);
   
    hipEventRecord(stopEvent,0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf(" %d  %f\n", i, time);

  }

  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
  hipFree(dA);
}

int main(int argc, char **argv)
{

  offsetExperiment();
  strideExperiment();

  
}