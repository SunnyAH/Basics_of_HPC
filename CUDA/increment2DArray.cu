#include "hip/hip_runtime.h"
%%cu

#include<stdio.h>

__device__ int dA[4][10];
int A[4][10];

void initializeHostArray()
{
    for(int i=0; i<4; i++)
     {
      for(int j=0; j<10; j++)
      {
          A[i][j] = i * 10 + j;
      }
     }
}
void printHostArray()
{
    for(int i=0; i<4; i++)
     {
      for(int j=0; j<10; j++)
      {
          printf("%d ",A[i][j]);
      }
      printf("\n");
     }
}
__global__ void increment()
{
    int r,c;
    c = blockIdx.x * blockDim.x + threadIdx.x;
    r = blockIdx.y * blockDim.y + threadIdx.y;
    dA[r][c]++;
}
int main()
{

     dim3 threadsPerBlock(5,2);
     dim3 blocksPerGrid(2,2);

     initializeHostArray();
     printHostArray();
     hipMemcpyToSymbol(HIP_SYMBOL(dA), A, 40*sizeof(int));
     
     increment<<<blocksPerGrid, threadsPerBlock>>>();
     
     hipMemcpyFromSymbol(A, HIP_SYMBOL(dA), 40*sizeof(int));
     printHostArray();
     hipDeviceSynchronize();
}