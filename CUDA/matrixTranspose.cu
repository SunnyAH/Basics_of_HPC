#include "hip/hip_runtime.h"
%%cu

#include<iostream>
#include<stdlib.h>
using namespace std;

#define BLOCK_SIZE 8
#define N 40

__device__ int dA[N][N];
int A[N][N];

void printMatrix(int n)
{
    int i,j;
 
  for(i=0; i<n; i++)
    {
        for(j=0; j<n; j++)
        {
            cout<<A[i][j]<<" "; 
        }
          cout<<endl;
    }    
}
    
__global__ void transposeUsingSharedMemory(int n) 
{

    int r,c,newr,newc, lr, lc;

  __shared__ int sA[BLOCK_SIZE][BLOCK_SIZE];
 
    r = blockIdx.y * blockDim.y + threadIdx.y;
    c = blockIdx.x * blockDim.x + threadIdx.x; 
    lc = threadIdx.x; lr = threadIdx.y;
    newr = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    newc = blockIdx.y * BLOCK_SIZE + threadIdx.x;

   
    sA[lc][lr]  = dA[r][c]; //dA[r * N + c];

    __syncthreads();

    //dA[newr * N + newc];
   dA[newr][newc] = sA[lr][lc];
        
}

int main()
{
    
    int n,i,j;
 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;
 
     n = N;
    

    dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE,1);
	  dim3 gridSize(N/BLOCK_SIZE,N/BLOCK_SIZE,1);
 
   
    for(i=0; i<n; i++)
    {
        for(j=0; j<n; j++)
        {
            A[i][j] = i*10 + j;
        }
          cout<<endl;
    }    
    hipMemcpyToSymbol(HIP_SYMBOL(dA),A,sizeof(int)*N*N);
 
    hipEventRecord(start,0);
    transposeUsingSharedMemory<<<gridSize,blockSize>>>(n);
    hipEventRecord(stop,0);
    hipEventSynchronize (stop); 
    hipEventElapsedTime (&time, start, stop); 
    
    hipEventDestroy (start);
    hipEventDestroy (stop);
 
    cout<<"Time taken: "<<time<<" milli seconds"<<endl;
 
 
    hipMemcpyFromSymbol(A,HIP_SYMBOL(dA), sizeof(int)*N*N);
  
  printMatrix(n);


    

}