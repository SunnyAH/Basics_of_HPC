#include "hip/hip_runtime.h"
%%cu



#include <stdio.h>
#include<iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024
#define N 500000 //50000

using namespace std;


__global__ void sum( int *A,  int stride)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i % (2*stride) == 0)
    {
        A[i] = A[i] + A[i+stride];
    }
}

int main(void)
{
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;
    int threadsPerBlock;
    int blocksPerGrid; 
     int n,stride;

    threadsPerBlock = BLOCK_SIZE;
    n = N;
 
    blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    
    hipError_t err = hipSuccess;


    int *A = (int *)malloc(n * sizeof(int));
    int result=0;

    for (int i = 0; i < n; ++i)
    {
        A[i] = i+1;
        result = result + A[i];
    }
    
    int *dA = NULL;
    err = hipMalloc((void **)&dA, n * sizeof(int));
   
    err = hipMemcpy(dA, A, n * sizeof(int), hipMemcpyHostToDevice);



    
    hipEventRecord(start,0);
    for(stride=1; stride < n; stride = stride * 2)
    {     
      sum<<<blocksPerGrid, threadsPerBlock>>>(dA, stride);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize (stop); 
    hipEventElapsedTime (&time, start, stop); 
    
    hipEventDestroy (start);
    hipEventDestroy (stop);
 
     cout<<"Time taken: "<<time<<" milli seconds"<<endl;
    
    err = hipMemcpy(A, dA, sizeof(int), hipMemcpyDeviceToHost);   
    hipDeviceSynchronize();
    cout<<A[0]<<endl;
    cout<<result<<endl;
    
    err = hipFree(dA);    

    free(A);
    

    return 0;
}