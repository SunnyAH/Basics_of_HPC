#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#define N 32

int A[N],count;
__device__ int dA[N], dCount;

void initHostArray()
{
    for(int i=0; i<N;i++)
    {
        A[i] = i;
    }
}
void printHostArray()
{
    for(int i=0; i<N;i++)
    {
        printf("%d \n",A[i]);
    }
}

__global__ void countEven()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
 
    if(dA[i]%2==0)
    {
     atomicAdd(&dCount,1);
    }
}

int main()
{

 initHostArray();
 hipMemcpyToSymbol(HIP_SYMBOL(dA),A,sizeof(int)*N);
 countEven<<<1,N>>>();
 hipMemcpyFromSymbol( &count, HIP_SYMBOL(dCount), sizeof(int));
 hipDeviceSynchronize();
 printf("%d ",count);
 return 0;

}
